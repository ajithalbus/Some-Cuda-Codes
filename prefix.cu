#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

#include<iostream>
#include "kernel.h"
#define B 32
using namespace std;

int* prefix(int *x,int n){
int p=ceil(n*1.0/B);
;
int *S;
int *Aux;
int *S_host;
int *Aux_host;
S_host=(int *)malloc(sizeof(x[0])*(n+1));
Aux_host=(int *)malloc(sizeof(x[0]*p));
hipMalloc(&S,sizeof(x[0])*n);
hipMemcpy(S,x,sizeof(x[0])*n,hipMemcpyHostToDevice);

hipMalloc(&Aux,sizeof(x[0])*p);



for(int d=1;d<=ceil(log(B)+1);d++){
    k1<<<p,B>>>(S,d);
}



hipDeviceSynchronize();




k2<<<1,p>>>(Aux,S);

hipDeviceSynchronize();




for(int d=1;d<=ceil(log(p)+1);d++){
    k1<<<1,p>>>(Aux,d);
}
hipDeviceSynchronize();

/*hipMemcpy(Aux_host,Aux,sizeof(x[0])*p,hipMemcpyDeviceToHost);
for(int i=0;i<p;i++){
    cout<<i<<'-'<<Aux_host[i]<<"\n";
}
*/

k4<<<p,B-1>>>(Aux,S);

hipDeviceSynchronize();

k5<<<1,p>>>(Aux,S);

hipDeviceSynchronize();

hipMemcpy(S_host,S,sizeof(x[0])*n,hipMemcpyDeviceToHost);
/*
for(int i=0;i<n;i++){
    cout<<i+1<<'-'<<S_host[i]<<"\n";
}*/
hipFree(S);
hipFree(Aux);
free(Aux_host);
/*
hipMemcpy(S_host,S,sizeof(x[0])*n,hipMemcpyDeviceToHost);
for(int i=0;i<n;i++){
    cout<<i+1<<'-'<<S_host[i]<<"\n";
}*/

if(S_host[0]!=x[0]){
S_host[n]=S_host[n-1]+x[n];
return S_host+1;}
return S_host;

}