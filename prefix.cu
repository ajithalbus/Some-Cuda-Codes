#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<iostream>
#include "kernel.h"
#define B 32
using namespace std;

void prefix(int *x,int n){
int p=ceil(n*1.0/B);

int *S;
int *Aux;
int *S_host;

S_host=(int *)malloc(sizeof(x[0])*n);

hipMalloc(&S,sizeof(x[0])*n);
hipMemcpy(S,x,sizeof(x[0])*n,hipMemcpyHostToDevice);

hipMalloc(&Aux,sizeof(x[0])*p);

for(int d=1;d<=log(B);d++){
    k1<<<p,B>>>(S,d);
}
hipDeviceSynchronize();



k2<<<1,p>>>(Aux,S);

hipDeviceSynchronize();


for(int d=1;d<=ceil(log(p));d++){
    k3<<<1,p>>>(Aux,d);
}
hipDeviceSynchronize();

k4<<<p,B-1>>>(Aux,S);

hipDeviceSynchronize();

k5<<<1,p>>>(Aux,S);

hipDeviceSynchronize();
hipMemcpy(S_host,S,sizeof(x[0])*n,hipMemcpyDeviceToHost);
for(int i=0;i<n;i++){
    cout<<S_host[i]<<endl;
}

}