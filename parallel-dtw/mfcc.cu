#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include "mfcc.h"
#include<string>
using namespace std;
//-Xcompiler -fopenmp


     mfcc::mfcc(){
        cout<<"DEFAULT CONSTRUCTOR"<<endl;
    }
     mfcc::mfcc(string file_name){
        fstream file;
        file.open (file_name.c_str(), ios::in );
        file>>tmp>>N;
        features=(feature *)malloc(N*sizeof(feature));
        for(int i=0;i<N;i++){
            for(int j=0;j<38;j++){
                file>>features[i].x[j];
            }
        }
        //cout<<features[0].x[37];
    }

    



__host__ __device__ double euclids(feature a,feature b){
    int i;
    double value=0;
    for (i=0;i<38;i++){
        value+=(a.x[i]-b.x[i])*(a.x[i]-b.x[i]);
        //printf("%f-%f=%f\n",a.x[i],b.x[i],value);;
    }
    return sqrt(value);
}