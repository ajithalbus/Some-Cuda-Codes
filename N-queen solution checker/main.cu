#include <thrust/pair.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <cstdlib>
#include <iostream>
using namespace std;

void print(const thrust::device_vector< thrust::pair<int,int> > &v)
{thrust::pair<int,int> tmp;
  for(size_t i = 0; i < v.size(); i++)
	{tmp=v[i];
    std::cout << " " << tmp.first <<tmp.second;
  std::cout << "\n";}
}
__device__ int device_flag=0;

struct funct {
  funct(thrust::pair<int,int>  x) : x(x) {}
__device__   
void operator()(thrust::pair<int,int>  y) const {
	//printf("pair-%d-%d,%d-%d\n",x.first,x.second,y.first,y.second);
	if(device_flag==1) return;
	if(x.first==y.first||x.second==y.second||x.first-y.first==x.second-y.second||x.first-y.first==-(x.second-y.second)) {printf("NO\n%d %d\n%d %d\n",x.first,x.second,y.first,y.second);
		device_flag=1;
		
return;
	}

	

}

private:
  thrust::pair<int,int>  x;
	
};



int main(void)
{
int n,i,j,tmp,ptr=0;
cin>>n;
thrust::pair<int,int> pairs[n];
for(i=0;i<n;i++)
for(j=0;j<n;j++)
{cin>>tmp;
if(tmp==1){
pairs[ptr].first=i;
pairs[ptr].second=j;
ptr++;}
}
thrust::device_vector< thrust::pair<int,int> > q_pos(pairs,pairs+n);

int host_flag=0;

for(i=0;i<n;i++) //like a nested for
{
thrust::for_each(q_pos.begin()+i+1,q_pos.end(),funct(q_pos[i]));
hipMemcpyFromSymbol((void*)&host_flag,device_flag,sizeof(int));
if(host_flag==1) break;
}

if(host_flag==0) cout<<"YES\n";
//print(q_pos);
  return 0;
}
