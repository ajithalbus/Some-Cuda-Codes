#include<stdio.h>
#include<hip/hip_runtime.h>

#define B 32
__device__ int pow(int a,int b){
    int result=1;
    while(b-->0){
        result*=a;
    }
    return result;
}

__device__ int d_min(int a,int b){
    if(a<b) return a;
    return b;
}

__device__ int d_max(int a,int b){
    if(a>b) return a;
    return b;
}

__device__ int dist(int a,int b){
    return max(a,b)-min(a,b);
}

__global__ void k1(int *a,int d){
   int tid=blockIdx.x*B+threadIdx.x;
   if(threadIdx.x>=pow(2,d-1)){
       a[tid]+=a[tid-pow(2, d-1)];
   }
}

__global__ void k2(int *Aux,int *S){

Aux[threadIdx.x]=S[(threadIdx.x+1)*B-1];
}

__global__ void k3(int *Aux,int d){
    if(threadIdx.x>=pow(2,d)){
        Aux[threadIdx.x]+=Aux[threadIdx.x-pow(2,d-1)];
    }
}

__global__ void k4(int *Aux,int *S){
    if(blockIdx.x==0) return;
    int tid=blockIdx.x*B+threadIdx.x;
    S[tid]+=Aux[blockIdx.x-1];
}

__global__ void k5(int *Aux,int *S){
    if(threadIdx.x==0) return;
    S[(threadIdx.x+1)*B-1]=Aux[threadIdx.x];
}

__global__ void q(int i,int *D,int *x_device,int *y_device,int *X,int *Y,int xN,int yN){
    int j=threadIdx.x,t,z;
    if(j==0) {X[0]=x_device[0];return;}

    t=d_min(D[(i-1)*xN+j],D[(i-1)*xN+(j-1)])+dist(y_device[i],x_device[j]);
    z=t-Y[j];
    X[j]=min(z,X[j-1]);
    printf("x-%d y-%d\n",X[j],z);
    D[i*xN+j]=X[j]+Y[j];

}