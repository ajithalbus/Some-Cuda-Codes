#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "mfcc.h"
#define B 32
__device__ int pow(int a,int b){
    int result=1;
    while(b-->0){
        result*=a;
    }
    return result;
}

__device__ int d_min(int a,int b){
    if(a<b) return a;
    return b;
}

__device__ double d_min(double a,double b){
    if(a<b) return a;
    return b;
}

__host__ __device__ double euclid(feature a,feature b){
    int i;
    double value=0;
    for (i=0;i<38;i++){
        value+=(a.x[i]-b.x[i])*(a.x[i]-b.x[i]);
        //printf("%f-%f=%f\n",a.x[i],b.x[i],value);;
    }
    return sqrt(value);
}


__device__ int d_max(int a,int b){
    if(a>b) return a;
    return b;
}

__device__ int dist(int a,int b){
    return max(a,b)-min(a,b);
}

__global__ void k1(int *a,int d){
   int tid=blockIdx.x*B+threadIdx.x;
   if(threadIdx.x>=pow(2,d-1)){
       a[tid]+=a[tid-pow(2, d-1)];
   }
}

__global__ void k2(int *Aux,int *S){

Aux[threadIdx.x]=S[(threadIdx.x+1)*B-1];
}

__global__ void k3(int *Aux,int d){
    if(threadIdx.x>=pow(2,d)){
        Aux[threadIdx.x]+=Aux[threadIdx.x-pow(2,d-1)];
    }
}

__global__ void k4(int *Aux,int *S){
    if(blockIdx.x==0) return;
    int tid=blockIdx.x*B+threadIdx.x;
    S[tid]+=Aux[blockIdx.x-1];
}

__global__ void k5(int *Aux,int *S){
    if(threadIdx.x==0) return;
    S[(threadIdx.x+1)*B-1]=Aux[threadIdx.x];
}

__global__ void q(int i,int *D,int *x_device,int *y_device,int *X,int *Y,int xN,int yN){
    int j=threadIdx.x,t,z;
    if(j==0) {X[0]=x_device[0];return;}

    t=d_min(D[(i-1)*xN+j],D[(i-1)*xN+(j-1)])+dist(y_device[i],x_device[j]);
    z=t-Y[j];
    X[j]=min(z,X[j-1]);
    //printf("x-%d y-%d\n",X[j],z);
    D[i*xN+j]=X[j]+Y[j];

}

__global__ void q2(int i,int *D,int *x_device,int *y_device,int xN,int yN){
    int tid=threadIdx.x;
    if(tid==0){
        D[i*xN]=dist(y_device[i],x_device[tid])+D[(i-1)*xN];
    }    
    else {
        D[i*xN+tid]=d_min(D[(i-1)*xN+tid],D[(i-1)*xN+(tid-1)])+dist(y_device[i],x_device[tid]);
    }
}

__global__ void q3(int i,double *D,feature *x_device,feature *y_device,int xN,int yN){
    
    int tid=threadIdx.x;
    if(tid==0){
        D[i*xN]=euclid(y_device[i],x_device[tid])+D[(i-1)*xN];
    }    
    else {
        D[i*xN+tid]=d_min(D[(i-1)*xN+tid],D[(i-1)*xN+(tid-1)])+euclid(y_device[i],x_device[tid]);
    }
}