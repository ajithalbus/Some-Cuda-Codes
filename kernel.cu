#include<stdio.h>
#include<hip/hip_runtime.h>

#define B 32
__device__ int pow(int a,int b){
    int result=1;
    while(b-->0){
        result*=a;
    }
    return result;
}

__global__ void k1(int *a,int d){
   int tid=blockIdx.x*B+threadIdx.x;
   if(tid>=pow(2,d)){
       a[tid]+=a[tid-pow(2, d-1)];
   }
}

__global__ void k2(int *Aux,int *S){

Aux[threadIdx.x]=S[(threadIdx.x+1)*B-1];
}

__global__ void k3(int *Aux,int d){
    if(threadIdx.x>=pow(2,d)){
        Aux[threadIdx.x]+=Aux[threadIdx.x-pow(2,d-1)];
    }
}

__global__ void k4(int *Aux,int *S){
    if(blockIdx.x==0) return;
    int tid=blockIdx.x*B+threadIdx.x;
    S[tid]+=Aux[blockIdx.x-1];
}

__global__ void k5(int *Aux,int *S){
    if(threadIdx.x==0) return;
    S[(threadIdx.x+1)*B-1]=Aux[threadIdx.x];
}