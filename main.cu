#include "hip/hip_runtime.h"
#include<stdio.h>
#include "prefix.h"
#include "dtw.h"
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <boost/filesystem.hpp>
#include <string>
#include "mfcc.h"
#include <omp.h>
using namespace std;
using namespace boost::filesystem;

//mfcc

//mfcc end
char * to_string(int x){
    char *a=(char *)malloc(2);
    a[0]=x+48;
    a[1]='\0';
    return a;
}

int argmin(double v[],int n){
    int i,min=0;
    double minval=1000000.0;
    for(i=0;i<n;i++){
        if(v[i]<minval){
            minval=v[i];
            //cout<<v[i]<<' ';
            min=i+1;
        }
    }
    return min;
}

int argmin_val(double v[],int n){
    int i,min=0;
    double minval=1000000.0;
    for(i=0;i<n;i++){
        if(v[i]<minval){
            minval=v[i];
            //cout<<v[i]<<' ';
            min=i+1;
        }
    }
    return minval;
}

int main(){
    /*int x[]={1,2,3,4,5},y[]={2,4,6,2};
    int d=dtw_nv(x,y,5,4);
    //cout<<d;
    
    */
    int i,j;
    string train="/home/ganesh/nw/dtw_cuda/datas/";
    
    string test="/home/ganesh/nw/dtw_cuda/datas_test/";
    string testn;
    cout<<"Enter test file :: ";
    cin>>testn;
    
    mfcc testmfcc(test+testn);

    mfcc* templates=(mfcc *)malloc(sizeof(mfcc)*30);
    //omp_set_num_threads(30);
    double d[10];
    for(i=1;i<=9;i++){
        double temp_d[30];
        
        for(j=0;j<30;j++){
            templates[j]=mfcc(train+to_string(i)+"/"+to_string(j)+".mfcc");
        }

        #pragma omp parallel for
        for(j=0;j<30;j++){
            temp_d[j]=dtw_nv(testmfcc,templates[j]);
        }
        #pragma omp barrier
        #pragma omp flush
        
        d[i-1]=argmin_val(temp_d,30);
    }

    cout<<endl;
    /*for(i=0;i<9;i++)
    cout<<"diff-score - "<<i+1<<'-'<<d[i]<<endl;
*/
    cout<<"Predicted label :: "<<argmin(d,9)<<endl;
    
}