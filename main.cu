#include "hip/hip_runtime.h"
#include<stdio.h>
#include "prefix.h"
#include "dtw.h"
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <boost/filesystem.hpp>
#include <string>
//#include "mfcc.h"

using namespace std;
using namespace boost::filesystem;

//mfcc

class feature{
public:
    float x[38];

};


class mfcc{

public:
    int N,tmp;
    feature *features;
    __host__ mfcc(){
        cout<<"DEFAULT CONSTRUCTOR"<<endl;
    }
    __host__ mfcc(string file_name){
        fstream file;
        file.open (file_name.c_str(), ios::in );
        file>>tmp>>N;
        features=(feature *)malloc(N*sizeof(feature));
        for(int i=0;i<N;i++){
            for(int j=0;j<38;j++){
                file>>features[i].x[j];
            }
        }
        //cout<<features[0].x[37];
    }

    

};


__host__ __device__ float euclids(feature *a,feature *b){
    int i;
    float value=0;
    for (i=0;i<38;i++){
        value+=(a->x[i]-b->x[i])*(a->x[i]-b->x[i]);
    }
    return sqrt(value);
}
//mfcc end



int main(){
    //int x[]={1,2,3};
    /*for(int i=1;i<=5;i++){
        x[i]=i;
    }*/
    //int d=dtw_nv(x,x,3,3);
    string fd="/home/ganesh/nw/dtw_cuda/datas/1/ac_";
    
    mfcc one(fd+"1.mfcc");
    mfcc two(fd+"2.mfcc");
    mfcc three(fd+"3.mfcc");
    
}