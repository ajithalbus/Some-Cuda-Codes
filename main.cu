#include "hip/hip_runtime.h"
#include<stdio.h>
#include "prefix.h"
#include "dtw.h"
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <boost/filesystem.hpp>
#include <string>
#include "mfcc.h"
#include <omp.h>
using namespace std;
using namespace boost::filesystem;

//mfcc

//mfcc end
char * to_string(int x){
    char *a=(char *)malloc(2);
    a[0]=x+48;
    a[1]='\0';
    return a;
}

int argmin(double v[],int n){
    int i,min=0;
    double minval=1000000.0;
    for(i=0;i<n;i++){
        if(v[i]<minval){
            minval=v[i];
            //cout<<v[i]<<' ';
            min=i+1;
        }
    }
    return min;
}

int main(){
    /*int x[]={1,2,3,4,5},y[]={2,4,6,2};
    int d=dtw_nv(x,y,5,4);
    //cout<<d;
    
    */
    int i;
    string train="/home/ganesh/nw/dtw_cuda/datas/";
    
    string test="/home/ganesh/nw/dtw_cuda/datas/";
    string testn;
    cin>>testn;
    
    mfcc testmfcc(test+testn);
       mfcc* templates=(mfcc *)malloc(sizeof(mfcc)*10);
    
    #pragma omp parallel for 
    for(i=1;i<=9;i++){
        templates[i-1]=mfcc(train+to_string(i)+"/ac_"+to_string(i)+".mfcc");
        //cout<<"D-READ "<<i<<endl;
    }
    //cout<<train+to_string(1)+"/ac_"+to_string(1)+".mfcc";
    //cout<<dtw_nv(templates[0],testmfcc);
    
    double d[9];
    #pragma omp parallel for
    for(i=0;i<9;i++){
        d[i]=dtw_nv(testmfcc,templates[i]);
        //cout<<"D-TEST "<<i<<endl;
    }
    cout<<argmin(d,9);
      
}