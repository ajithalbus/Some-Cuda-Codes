#include "hip/hip_runtime.h"
#include<stdio.h>
#include "prefix.h"
#include "dtw.h"
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <boost/filesystem.hpp>
#include <string>
#include "mfcc.h"
#include <omp.h>
using namespace std;
using namespace boost::filesystem;

//mfcc

//mfcc end
char * to_string(int x){
    char *a=(char *)malloc(2);
    a[0]=x+48;
    a[1]='\0';
    return a;
}

int argmin(double v[],int n){
    int i,min=0;
    double minval=1000000.0;
    for(i=0;i<n;i++){
        if(v[i]<minval){
            minval=v[i];
            //cout<<v[i]<<' ';
            min=i+1;
        }
    }
    return min;
}

int main(){
    //int x[]={1,2,3};
    /*for(int i=1;i<=5;i++){
        x[i]=i;
    }*/
    //int d=dtw_nv(x,x,3,3);
    int i;
    string train="/home/ganesh/nw/dtw_cuda/datas/";
    
    string test="/home/ganesh/nw/dtw_cuda/datas/";
    string testn;
    cin>>testn;
    
    mfcc testmfcc(test+testn);
       mfcc* templates=(mfcc *)malloc(sizeof(mfcc)*10);
    
    #pragma omp parallel for 
    for(i=1;i<=9;i++){
        templates[i-1]=mfcc(train+to_string(i)+"/ac_"+to_string(i)+".mfcc");
        //cout<<"D-READ "<<i<<endl;
    }
    //cout<<train+to_string(1)+"/ac_"+to_string(1)+".mfcc";
    //cout<<dtw_nv(templates[0],testmfcc);
    
    double d[9];
    #pragma omp parallel for
    for(i=0;i<9;i++){
        d[i]=dtw_nv(testmfcc,templates[i]);
        //cout<<"D-TEST "<<i<<endl;
    }
    cout<<argmin(d,9);
    /*
        mfcc one(train+"1/ac_1.mfcc");

    mfcc two(train+"2/ac_2.mfcc");
    mfcc three(train+"3/ac_3.mfcc");
    string testn;
    
    double d[3];
    
    d[0]=dtw_nv(one,testmfcc);
    d[1]=dtw_nv(two,testmfcc);
    d[2]=dtw_nv(three,testmfcc);

    cout<<argmin(d,3);

    */

    //cout<<d;  
}