#include "hip/hip_runtime.h"
#include<stdio.h>
#include "prefix.h"
#include "dtw.h"
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <boost/filesystem.hpp>
#include <string>
#include "mfcc.h"

using namespace std;
using namespace boost::filesystem;

//mfcc

//mfcc end

int argmin(double v[],int n){
    int i,min=0;
    double minval=1000000.0;
    for(i=0;i<n;i++){
        if(v[i]<minval){
            minval=v[i];
            //cout<<v[i]<<' ';
            min=i+1;
        }
    }
    return min;
}

int main(){
    //int x[]={1,2,3};
    /*for(int i=1;i<=5;i++){
        x[i]=i;
    }*/
    //int d=dtw_nv(x,x,3,3);
    string train="/home/ganesh/nw/dtw_cuda/datas/";
    
    string test="/home/ganesh/nw/dtw_cuda/datas/";
    
    mfcc one(train+"1/ac_1.mfcc");

    mfcc two(train+"2/ac_2.mfcc");
    mfcc three(train+"3/ac_3.mfcc");
    string testn;
    cin>>testn;
    
    mfcc testmfcc(test+testn);
    
    double d[3];
    
    d[0]=dtw_nv(one,testmfcc);
    d[1]=dtw_nv(two,testmfcc);
    d[2]=dtw_nv(three,testmfcc);

    cout<<argmin(d,3);

    

    //cout<<d;  
}