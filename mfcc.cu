#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include "mfcc.h"
using namespace std;
//-Xcompiler -fopenmp


    __host__ mfcc::mfcc(){
        cout<<"DEFAULT CONSTRUCTOR"<<endl;
    }
    __host__ mfcc::mfcc(char file_name[]){
        fstream file;
        file.open (file_name, ios::in );
        file>>tmp>>N;
        features=(feature *)malloc(N*sizeof(feature));
        for(int i=0;i<N;i++){
            for(int j=0;j<38;j++){
                file>>features[i].x[j];
            }
        }
        //cout<<features[0].x[37];
    }

    



__host__ __device__ float euclids(feature *a,feature *b){
    int i;
    float value=0;
    for (i=0;i<38;i++){
        value+=(a->x[i]-b->x[i])*(a->x[i]-b->x[i]);
    }
    return sqrt(value);
}