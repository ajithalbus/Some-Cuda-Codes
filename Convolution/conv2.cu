#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define W 1024 // H/W
#define OW (W-4) //output H/W
#define D 8   //depth
#define T 5  //size of kern
#define N 32 //number of kerns
void fillMatrix(char *matrix){

char (*m)[W][D]=(char (*)[W][D])matrix;

for(int i=0;i<W;i++){
	for(int j=0;j<W;j++){
		for(int k=0;k<D;k++){
			m[i][j][k]=(i*j+j*k+i*k+i*2+j*3+k*4)%255;
				}
			}
		}
}


void fillTile(float *tile){

float (*t)[T][T][D]=(float (*)[T][T][D])tile;

for(int i=0;i<N;i++){
	for(int j=0;j<T;j++){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
			t[i][j][k][l]=fmod(-(i+1)*2.1+(j+1)*3.2-(k+1)*4.8+(l+1)*7.1,1.0);
				}
			}
		}
	}

}


void print_matrix_to_file2(float *m)//,unsigned , unsigned T,unsigned G, unsigned D)
 



{
	float (*mat)[T][T][D]=(float (*)[T][T][D])m;	
	const char *fname = "filter";
	FILE *f = fopen(fname, "w");

	

		for(unsigned i=0; i < N; i++) {
		for(unsigned j=0; j < T; j++)
		for(unsigned k=0; k < T; k++) 
		for(unsigned l=0; l < D; l++)
			fprintf(f,"%4f ", mat[i][j][k][l]);
		fprintf(f,"\n");
	}
	fclose(f);
}


void print_matrix_to_file3(float *m)//, unsigned numRows, unsigned numCols) {
	{const char *fname = "result";
	FILE *f = fopen(fname, "w");

	float (*mat)[OW][OW]=(float (*)[OW][OW])m;		

	for(unsigned i=0; i < N; i++) {
		for(unsigned j=0; j < OW; j++)
		for(unsigned k=0;k<OW;k++)
			fprintf(f,"%4f ", mat[i][j][k]);
		fprintf(f,"\n");
	}
	fclose(f);
}

void print_matrix_to_file1(char *m, unsigned numRows, unsigned numCols,unsigned d) {
	const char *fname = "mat";
	FILE *f = fopen(fname, "w");

	char (*mat)[numCols][d]=(char (*)[numCols][d])m;
	for(unsigned i=0; i < numRows; i++) {
		for(unsigned j=0; j < numCols; j++)
			for(unsigned k=0; k < d; k++)
				fprintf(f,"%d ", mat[i][j][k]);
		fprintf(f,"\n");
	}
	fclose(f);
}


/*
__global__ void convold(int *matrix,float *tile,float *output){

int filter=blockIdx.x;
int eX=blockIdx.y;
int eY=threadIdx.x;

int (*m)[W][W][D]=(int (*)[W][W][D])matrix;
float (*t)[N][T][T][D]=(float (*)[N][T][T][D])tile;
float (*o)[N][OW][OW]=(float (*)[N][OW][OW])output;

__shared__ int slice[W][D];

float psum;

if(eX<2||eX>W-3) return;

for(int j=0;j<T;j++){
	for(int i=0;i<D;i++){
		slice[eY][i]=matrix[(eX+j-2)*W+eY*W+i];
		
	}
__syncthreads();
	psum=0.0f;
	if(!(eY<2||eY>W-3)){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
				psum+=tile[filter*N+j*T+k*T+l]*slice[eY+k-2][l];	
				}
		}
		atomicAdd(&output[filter*N+(eX-2)*OW+eY-2],psum);
	}
__syncthreads();

}

}
*/

__global__ void conv(char *matrix,float *tile,float *output){

int filter=blockIdx.x;
int eX=blockIdx.y;
int eY=threadIdx.x;

char (*m)[W][D]=(char (*)[W][D])matrix;
float (*t)[T][T][D]=(float (*)[T][T][D])tile;
float (*o)[OW][OW]=(float (*)[OW][OW])output;

__shared__ int slice[W][D];

float psum;

if(eX<2||eX>W-3) return;

for(int j=0;j<T;j++){
	for(int i=0;i<D;i++){
		slice[eY][i]=m[(eX+j-2)][eY][i];
		
	}
__syncthreads();
	psum=0.0f;
	if(!(eY<2||eY>W-3)){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
				psum+=t[filter][j][k][l]*slice[eY+k-2][l];	
				}
		}
		atomicAdd(&o[filter][(eX-2)][eY-2],psum);
	}
__syncthreads();

}

}


int main()
{

char *matrix=(char*)malloc(sizeof(char)*W*W*D);
float *tile=(float*)malloc(sizeof(float)*T*T*D*N);
float *output=(float *)malloc(sizeof(float)*(N*OW*OW));


fillMatrix(matrix);
fillTile(tile);


char *Dmatrix;hipMalloc(&Dmatrix,sizeof(char)*W*W*D);
float *Dtile;hipMalloc(&Dtile,sizeof(float)*N*T*T*D);
float *Doutput;hipMalloc(&Doutput,sizeof(float)*(N*OW*OW));

hipMemcpy(Dmatrix, matrix, sizeof(char)*W*W*D,hipMemcpyHostToDevice);
hipMemcpy(Dtile, tile, sizeof(float)*T*T*D*N,hipMemcpyHostToDevice);

hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
float milliseconds = 0;

hipEventRecord(start);


conv<<<dim3(N,W),W>>>(Dmatrix,Dtile,Doutput);
hipDeviceSynchronize();


hipEventRecord(stop);
hipEventSynchronize(stop);
hipEventElapsedTime(&milliseconds, start, stop);
printf("%f\n",milliseconds);



hipMemcpy(output, Doutput, sizeof(float)*(N*OW*OW),hipMemcpyDeviceToHost);

/*
for(int i=0;i<N;i++){
	for(int j=0;j<T;j++){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
			printf("%.1f ",tile[i*N+j*T+k*T+l]);//=(i*j-j*k+i*k-k*l)%5;
				}
			}
		}
	}



for(int i=0;i<W;i++){
	for(int j=0;j<W;j++){
		for(int k=0;k<D;k++){
			//for(int l=0;l<N;l++){
			printf("%d ",matrix[i*W+j*W+k]);//=(i*j-j*k+i*k-k*l)%5;
			//	}
			}
		}
	}



printf("\n\n");
for(int i=0;i<N;i++){
	for(int j=0;j<OW;j++){
		for(int k=0;k<OW;k++){
			//for(int l=0;l<N;l++){
			printf("%.1f ",output[i*N+j*OW+k]);//=(i*j-j*k+i*k-k*l)%5;
			//	}
			}
		}
	}
*/

print_matrix_to_file1(matrix,W,W,D);
print_matrix_to_file2(tile);//,N,T,T,D);
print_matrix_to_file3(output);//,N,OW,OW);
}
