/*
Template code for convolution. CS6023, IITM */
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define W 1024 // Input DIM
#define OW (W-4) // Output DIM
#define D 8   // Input and Kernel Depth
#define T 5  // Kernel DIM
#define N 128 // Number of kernels

void fillMatrix(unsigned char *matrix){

unsigned char (*m)[W][D]=(unsigned char (*)[W][D])matrix;

for(int i=0;i<W;i++){
	for(int j=0;j<W;j++){
		for(int k=0;k<D;k++){
			m[i][j][k]=(i*j+j*k+i*k+i*2+j*3+k*4)%255;
				}
			}
		}
}



void fillKernel(float *kernel){

float (*t)[T][T][D]=(float (*)[T][T][D])kernel;

for(int i=0;i<N;i++){
	for(int j=0;j<T;j++){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
			t[i][j][k][l]=fmod(-(i+1)*2.1+(j+1)*3.2-(k+1)*4.8+(l+1)*7.1,1.0);
				}
			}
		}
	}
}



void print_matrix_to_file(float *m){

	const char *fname = "assignment4_out";
	FILE *f = fopen(fname, "w");

	float (*mat)[OW][OW]=(float (*)[OW][OW])m;		

	for(unsigned i=0; i < N; i++) {
		for(unsigned j=0; j < OW; j++)
			for(unsigned k=0;k<OW;k++)
				fprintf(f,"%4f ", mat[i][j][k]);
		fprintf(f,"\n");
	}
	fclose(f);
}
__global__ void conv(unsigned char *matrix,float *tile,float *output){

int filter=blockIdx.x;
int eX=blockIdx.y;
int eY=threadIdx.x;

unsigned char (*m)[W][D]=(unsigned char (*)[W][D])matrix;
float (*t)[T][T][D]=(float (*)[T][T][D])tile;
float (*o)[OW][OW]=(float (*)[OW][OW])output;

__shared__ unsigned char slice[W][D];

float psum;

if(eX<2||eX>W-3) return;

for(int j=0;j<T;j++){
	for(int i=0;i<D;i++){
		slice[eY][i]=m[(eX+j-2)][eY][i];
		
	}
__syncthreads();
	psum=0.0f;
	if(!(eY<2||eY>W-3)){
		for(int k=0;k<T;k++){
			for(int l=0;l<D;l++){
				psum+=t[filter][j][k][l]*slice[eY+k-2][l];	
				}
		}
		atomicAdd(&o[filter][(eX-2)][eY-2],psum);
	}
__syncthreads();

}

}



int main()
{

	unsigned char *matrix=(unsigned char*)malloc(sizeof(unsigned char)*W*W*D);
	float *kernel=(float*)malloc(sizeof(float)*T*T*D*N);
	float *output=(float *)malloc(sizeof(float)*N*OW*OW);


	fillMatrix(matrix);
	fillKernel(kernel);


	unsigned char *Dmatrix;hipMalloc(&Dmatrix,sizeof(unsigned char)*W*W*D);
	float *Dkernel;hipMalloc(&Dkernel,sizeof(float)*N*T*T*D);
	float *Doutput;hipMalloc(&Doutput,sizeof(float)*N*OW*OW);

	hipMemcpy(Dmatrix, matrix, sizeof(unsigned char)*W*W*D,hipMemcpyHostToDevice);
	hipMemcpy(Dkernel, kernel, sizeof(float)*T*T*D*N,hipMemcpyHostToDevice);


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	hipEventRecord(start,0);

	//Make your cuda kernel call
	
	conv<<<dim3(N,W),W>>>(Dmatrix,Dkernel,Doutput);
	hipDeviceSynchronize();


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("%f\n",milliseconds);


	hipMemcpy(output, Doutput, sizeof(float)*N*OW*OW,hipMemcpyDeviceToHost);

	//Use print_matrix_to_file function only 
	
	print_matrix_to_file(output);

}
