// Tested working with single block , in GPGPU-SIM with sm_20.
#include<stdio.h>
#include<math_functions.h>
#include<hip/hip_runtime.h>

/* The kernel "msort" should sort the input array using parallel merge-sort. */

__device__ int pow(int a,int b){
    int pwr=1,i;
    for(i=0;i<b;i++)
        pwr*=a;
    return pwr;
}

__global__ void msort(int *d_input, int* d_temp, int N)
{int lvl=1;
int id=threadIdx.x+blockIdx.x*blockDim.x;
int i,m,r,k,j,p;


while(int(pow(2,lvl))<=pow(2,int(ceil(log2(float(N))))))

{
    if(id<N){
    
if(id%pow(2,lvl)==0)
{
i=id;
m=(id+int(pow(2,lvl-1))>N)?N:id+int(pow(2,lvl-1));
j=m;
r=(id+int(pow(2,lvl))>N)?N:id+int(pow(2,lvl));
k=id;

for(p=id;p<r;p++)
d_temp[p]=d_input[p];

while(i<m &&j<r){
    if(d_temp[i]<d_temp[j]){
        d_input[k++]=d_temp[i++];
    }
    else
    {
        d_input[k++]=d_temp[j++];
    }

}
while(i<m){
    d_input[k++]=d_temp[i++];
}
while(j<r){
    d_input[k++]=d_temp[j++];
}

}


    
    }
    
lvl++;
__syncthreads();    


}
}





