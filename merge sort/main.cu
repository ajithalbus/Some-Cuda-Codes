#include<stdio.h>
#include "kernels.h"
#define N 32
int main()

{int tmpx[N];
//int x[N]={64, 63, 62, 61, 60, 59, 58, 57, 56, 55, 54, 53, 52, 51, 50, 49, 48, 47, 46, 45, 44, 43, 42, 41, 40, 39, 38, 37, 36, 35, 34, 33, 32, 31, 30, 29, 28, 27, 26, 25, 24, 23, 22, 21, 20, 19, 18, 17, 16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
int x[N]={32, 31, 30, 29, 28, 27, 26, 25, 24, 23, 22, 21, 20, 19, 18, 17, 16, 15, 14, 13, 12, 11, 10, 9, 8, 7, 6, 5, 4, 3, 2, 1};
int *kerx,*tmp,i;//*counter,c=99;
hipMalloc(&kerx,sizeof(int)*N);
hipMalloc(&tmp,sizeof(int)*N);
//hipMalloc(&counter,sizeof(int));
hipMemcpy(kerx,x,sizeof(int)*N,hipMemcpyHostToDevice);


msort<<<1,N>>>(kerx,tmp,N);

hipDeviceSynchronize();
hipMemcpy(x,kerx,sizeof(int)*N,hipMemcpyDeviceToHost);
hipMemcpy(tmpx,tmp,sizeof(int)*N,hipMemcpyDeviceToHost);

//hipMemcpy(&c,counter,sizeof(int),hipMemcpyDeviceToHost);
//printf("\ncounter-%d\n",c);


for(i=0;i<N;i++)
printf("%d\n",x[i]);


}
