#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "prefix.h"
#include "kernel.h"
#include<iostream>
#include "dtw.h"
using namespace std;

int abs(int a){
    return (a>0)?a:-a;
}

double dtw_nv(mfcc x,mfcc y){

    size_t ds=sizeof(x.features[0]);
    double *D;
    double *D_host;
    feature *x_device,*y_device;
    int i,j;
    

    D_host=(double *)malloc(ds*x.N*(y.N+1));
    
    hipMalloc(&D,ds*x.N*(y.N+1));
    //hipMemset(&D,0,ds*xN*(yN+1));
    
    hipMalloc(&x_device,ds*x.N);
    hipMalloc(&y_device,ds*y.N);
    
    hipMemcpy(x_device,x.features,ds*x.N,hipMemcpyHostToDevice);
    hipMemcpy(y_device,y.features,ds*y.N,hipMemcpyHostToDevice);
    
    //calculating 1st row & 1st col
    double tmp[x.N];
    tmp[0]=abs(euclids(x.features[0],y.features[0]));
    
    for(i=1;i<x.N;i++){
    tmp[i]=tmp[i-1]+abs(euclids(x.features[0],y.features[0]));
    }
    
    hipMemcpy(D,tmp,sizeof(double)*x.N,hipMemcpyHostToDevice);
    
    for(i=1;i<y.N;i++)
    q3<<<1,x.N>>>(i,D,x_device,y_device,x.N,y.N);


    
    
    hipMemcpy(D_host,D,ds*x.N*(y.N+1),hipMemcpyDeviceToHost);
    
    /*
    for(i=0;i<y.N;i++){
        for(j=0;j<x.N;j++){
            cout<<D_host[(i*x.N)+j]<<' ';
            
        }
        cout<<endl;
    }
    */
    //return 0;
    return D_host[(y.N-1)*x.N+(x.N-1)];
    
}

int dtw_nv(int *x,int *y,int xN,int yN){
    size_t ds=sizeof(x[0]);
    int *D;
    int *D_host;
    int *x_device,*y_device;
    int i,j;
    

    D_host=(int *)malloc(ds*xN*(yN+1));
    
    hipMalloc(&D,ds*xN*(yN+1));
    hipMemset(&D,0,ds*xN*(yN+1));
    
    hipMalloc(&x_device,ds*xN);
    hipMalloc(&y_device,ds*yN);
    
    hipMemcpy(x_device,x,ds*xN,hipMemcpyHostToDevice);
    hipMemcpy(y_device,y,ds*yN,hipMemcpyHostToDevice);
    
    //calculating 1st row & 1st col
    int tmp[xN];
    tmp[0]=abs(x[0]-y[0]);
    
    for(i=1;i<xN;i++){
    tmp[i]=tmp[i-1]+abs(x[i]-y[0]);
    }
    
    hipMemcpy(D,tmp,ds*xN,hipMemcpyHostToDevice);
    
    for(i=1;i<yN;i++)
    q2<<<1,xN>>>(i,D,x_device,y_device,xN,yN);


    
    
    hipMemcpy(D_host,D,ds*xN*(yN+1),hipMemcpyDeviceToHost);
    
    for(i=0;i<yN;i++){
        for(j=0;j<xN;j++){
            cout<<D_host[(i*xN)+j]<<' ';
            
        }
        cout<<endl;
    }
    
    return 0;
        
}


int dtw(int *x,int *y,int xN,int yN)
{
    size_t ds=sizeof(x[0]);
int *D,*y_host,*X,*Y;
int *D_host;
int *x_device,*y_device;
int i,j;

D_host=(int *)malloc(ds*xN*(yN+1));

hipMalloc(&D,ds*xN*(yN+1));
hipMalloc(&X,ds*xN);
hipMalloc(&Y,ds*xN);
hipMalloc(&x_device,ds*xN);
hipMalloc(&y_device,ds*yN);

hipMemcpy(x_device,x,ds*xN,hipMemcpyHostToDevice);
hipMemcpy(y_device,y,ds*yN,hipMemcpyHostToDevice);

int tmp[xN];
tmp[0]=abs(x[0]-y[0]);
for(i=1;i<xN;i++){
tmp[i]=tmp[i-1]+abs(x[i]-y[0]);
}

hipMemcpy(D,tmp,ds*xN,hipMemcpyHostToDevice);


for(i=1;i<yN;i++){
    hipMemcpy(D_host,D+(i-1)*xN,ds*xN,hipMemcpyDeviceToHost);
    y_host=prefix(D_host,xN);
    //for(int k=0;k<xN;k++) {cout<<"pre-"<<y_host[k];}
    hipMemcpy(Y,y_host,ds*xN,hipMemcpyHostToDevice);
    q<<<1,xN>>>(i,D,x_device,y_device,X,Y,xN,yN);
    hipDeviceSynchronize();
}

hipMemcpy(D_host,D,ds*xN*(yN+1),hipMemcpyDeviceToHost);

for(i=0;i<yN;i++){
    for(j=0;j<xN;j++){
        cout<<D_host[(i*xN)+j]<<' ';
        
    }
    cout<<endl;
}

return 0;
}